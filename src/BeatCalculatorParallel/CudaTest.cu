#include "hip/hip_runtime.h"
#include <stdio.h>
#include <kiss_fftr.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__global__ void memAssign(int N, unsigned short* in, hipfftReal* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) 
        out[index] = (hipfftReal)in[index];
}

void cudaTest() {
    printf("Inside Cuda file\n");
}

int cudaFFT(unsigned short* sample, int size, kiss_fft_cpx* out) {

    // Assign Variables
    hipfftHandle plan;
    hipfftReal* deviceDataIn;
    hipfftComplex* deviceDataOut;
    unsigned short* deviceShortArray;

    // Malloc Variables 
    hipMalloc(&deviceDataIn, sizeof(hipfftReal) * size);
    hipMalloc(&deviceDataOut, sizeof(hipfftReal) * (size/2 + 1));
    hipMalloc(&deviceShortArray, sizeof(unsigned short) * size);

    // Copy memory over
    hipMemcpy(deviceShortArray, sample, size, hipMemcpyHostToDevice);

    // Run a Kernel to convert to the hipfftReal format
    int threadsPerBlock = 32;
    int blocks = size/threadsPerBlock + 1;
    memAssign<<<blocks, threadsPerBlock>>>(size, deviceShortArray, deviceDataIn);
    hipDeviceSynchronize();

    // Now run the fft
    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, deviceDataIn, deviceDataOut);

    // TODO: Get data back from GPU

    // Cleanup
    hipfftDestroy(plan);
    hipFree(deviceDataIn);
    hipFree(deviceDataOut);
    hipFree(deviceShortArray);

    return 0;
}
