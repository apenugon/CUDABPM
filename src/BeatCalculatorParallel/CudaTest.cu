#include "hip/hip_runtime.h"
#include <stdio.h>
#include <kiss_fftr.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__global__ void memAssign(int N, unsigned short* in, hipfftReal* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) 
        out[index] = (hipfftReal)in[index];
}

void cudaTest() {
    printf("Inside Cuda file\n");
}

int cudaFFT(unsigned short* sample, int size, kiss_fft_cpx* out) {

    // Assign Variables
    hipfftHandle plan;
    hipfftReal* deviceDataIn;
    hipfftComplex* deviceDataOut;
    hipfftComplex* hostDataOut;
    unsigned short* deviceShortArray;

    // Malloc Variables 
    hipMalloc(&deviceDataIn, sizeof(hipfftReal) * size);
    hipMalloc(&deviceDataOut, sizeof(hipfftReal) * (size/2 + 1));
    hipMalloc(&deviceShortArray, sizeof(unsigned short) * size);
    hostDataOut = (hipfftComplex*)malloc(sizeof(hipfftComplex) * (size/2+1));

    // Copy memory over
    hipMemcpy(deviceShortArray, sample, size, hipMemcpyHostToDevice);

    // Run a Kernel to convert to the hipfftReal format
    int threadsPerBlock = 32;
    int blocks = size/threadsPerBlock + 1;
    memAssign<<<blocks, threadsPerBlock>>>(size, deviceShortArray, deviceDataIn);
    hipDeviceSynchronize();

    // Now run the fft
    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, deviceDataIn, deviceDataOut);
    hipDeviceSynchronize();
    // Get data back from GPU
    hipMemcpy(hostDataOut, deviceDataOut, size/2+1, hipMemcpyDeviceToHost);

    // Copy to out
    for (int i = 0; i < size/2 + 1; i++) {
        out[i].r = hostDataOut[i].x;
        out[i].i = hostDataOut[i].y;
    }

    // Cleanup
    hipfftDestroy(plan);
    hipFree(deviceDataIn);
    hipFree(deviceDataOut);
    hipFree(deviceShortArray);

    return 0;
}
