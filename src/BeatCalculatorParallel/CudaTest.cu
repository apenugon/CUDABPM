#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <kiss_fftr.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#ifndef M_PI
#define M_PI 3.14159265358979324
#endif
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
  }
}

__global__ void memAssignShort(int N, unsigned short* in, hipfftReal* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        out[index] = in[index];
        //printf("CUDA: float: %f, short: %i\n", out[index], in[index]);
    }
}

__global__ void printData(int N, hipfftComplex* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) 
        printf("CUDA: Output FFT %f %f\n", out[index].x, out[index].y);
}

__global__ void memAssignFloat(int N, float* in, hipfftReal* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) 
        out[index] = (hipfftReal)in[index];
}

void cudaTestR() {
    printf("Starting Test of CUDA FFT \n");
    
    int size = 16; //use small sample for testing
    float *onesSample, *zerosSample, *sineSample;
    hipfftHandle plan;
    hipfftReal* fftDataIn;
    hipfftComplex* fftDataOut;
    float* testArrayOnes;
    float* testArrayZeros;
    float* testArraySine;
    float2* retArrayOnes;
    float2* retArrayZeros;
    float2* retArraySine;
    int i;

    printf("Mallocing \n");
    // Malloc Variables 
    gpuErrchk( hipMalloc(&fftDataIn, sizeof(hipfftReal) * size) );
    gpuErrchk( hipMalloc(&fftDataOut, sizeof(hipfftReal) * (size/2 + 1)) );
    gpuErrchk( hipMalloc(&testArrayOnes, sizeof(float) * size) );
    gpuErrchk( hipMalloc(&testArrayZeros, sizeof(float) * size) );
    gpuErrchk( hipMalloc(&testArraySine, sizeof(float) * size) );
    onesSample = (float*)malloc(size * sizeof(float));
    zerosSample = (float*)malloc(size * sizeof(float));
    sineSample = (float*)malloc(size * sizeof(float));
    retArrayOnes = (float2*)malloc((size/2 +1) * sizeof(float2));
    retArrayZeros = (float2*)malloc((size/2 +1) * sizeof(float2));
    retArraySine = (float2*)malloc((size/2 +1) * sizeof(float2));

    //Set up tests
    for (i = 0; i < size; i++) {
      onesSample[i] = 1;
      zerosSample[i] = 0;
      sineSample[i] = sin(2 * M_PI * 4 * i / size);
    }

    printf("Memcpy \n");
    // Copy memory over
    gpuErrchk( hipMemcpy(testArrayOnes, onesSample, size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(testArrayZeros, zerosSample, size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(testArraySine, sineSample, size, hipMemcpyHostToDevice) );


    //Testing wave of Ones
    printf("Run Ones Test \n");

    // Run a Kernel to convert to the hipfftReal format
    int threadsPerBlock = 32;
    int blocks = size/threadsPerBlock + 1;
    memAssignFloat<<<blocks, threadsPerBlock>>>(size, testArrayOnes, fftDataIn);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    
    // Now run the fft
    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, fftDataIn, fftDataOut);

    // Get data back from GPU
    gpuErrchk( hipMemcpy(retArrayOnes,fftDataOut, size/2 +1, hipMemcpyDeviceToHost) );

    //Testing wave of Zeros
    printf("Run Zeros Test \n");
    memAssignFloat<<<blocks, threadsPerBlock>>>(size, testArrayZeros, fftDataIn);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, fftDataIn, fftDataOut);

    gpuErrchk( hipMemcpy(retArrayZeros,fftDataOut, size/2 +1, hipMemcpyDeviceToHost) );
   
    //Testing sine wave 
    printf("Run Sine Test \n");

    memAssignFloat<<<blocks, threadsPerBlock>>>(size, testArraySine, fftDataIn);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, fftDataIn, fftDataOut);

    gpuErrchk( hipMemcpy(retArraySine,fftDataOut, size/2 +1, hipMemcpyDeviceToHost) );
    
    //Ones Data
    printf("Ones Result\n");
    for (i = 0; i < size; i++) {
      printf(" in[%2zu] = %+f    ", i, onesSample[i]);
      if (i < size / 2 + 1)
        printf("out[%2zu] = %+f , %+f", i, retArrayOnes[i].x, retArrayOnes[i].y);
      printf("\n");
    }  
    
    //Zeros Data
    printf("Ones Result\n");
    for (i = 0; i < size; i++) {
      printf(" in[%2zu] = %+f    ", i, zerosSample[i]);
      if (i < size / 2 + 1)
        printf("out[%2zu] = %+f , %+f", i, retArrayZeros[i].x, retArrayZeros[i].y);
      printf("\n");
    }  
    
    //Sine Data
    printf("Ones Result\n");
    for (i = 0; i < size; i++) {
      printf(" in[%2zu] = %+f    ", i, sineSample[i]);
      if (i < size / 2 + 1)
        printf("out[%2zu] = %+f , %+f", i, retArraySine[i].x, retArraySine[i].y);
      printf("\n");
    } 
 
    // Cleanup CUDA Side
    hipfftDestroy(plan);
    hipFree(fftDataIn);
    hipFree(fftDataOut);
    hipFree(testArrayOnes);
    hipFree(testArrayZeros);
    hipFree(testArraySine);
    
    //Free arrays
    free(onesSample);
    free(zerosSample);
    free(sineSample);
    free(retArrayOnes);
    free(retArrayZeros);
    free(retArraySine);

}

void cudaTest() {
  printf("Inside CUDA File\n");

}

int cudaFFT(unsigned short* sample, int size, kiss_fft_cpx* out) {

    // Assign Variables
    hipfftHandle plan;
    hipfftReal* deviceDataIn;
    hipfftComplex* deviceDataOut;
    hipfftComplex* hostDataOut;
    unsigned short* deviceShortArray;

    // Malloc Variables 
    hipMalloc(&deviceDataIn, sizeof(hipfftReal) * size);
    hipMalloc(&deviceDataOut, sizeof(hipfftComplex) * (size/2 + 1));
    hipMalloc(&deviceShortArray, sizeof(unsigned short) * size);
    hostDataOut = (hipfftComplex*)malloc(sizeof(hipfftComplex) * (size/2+1));
    if (hipGetLastError() != hipSuccess) {
        printf("Failed to allocate stuff on GPU\n");
        return 0;
    }

       // Copy memory over
     if (hipMemcpy(deviceShortArray, sample, size * sizeof(unsigned short), hipMemcpyHostToDevice) != hipSuccess) {
         printf("Failed to copy shorts over\n");
         return 0;
     }

    // Run a Kernel to convert to the hipfftReal format
    int threadsPerBlock = 512;
    int blocks = (size + threadsPerBlock - 1)/threadsPerBlock;
    memAssignShort<<<blocks, threadsPerBlock>>>(size, deviceShortArray, deviceDataIn);
    hipDeviceSynchronize();

    // Now run the fft
    if (hipfftPlan1d(&plan, size, HIPFFT_R2C, 1) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - plan creation failed\n");
        return 0;
    }
    if (hipfftExecR2C(plan, deviceDataIn, deviceDataOut) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - execution of FFT failed\n");
        return 0;
    }
    if (hipDeviceSynchronize() != hipSuccess) {
        printf("Failed to sync\n");
        return 0;
    }
    // Get data back from GPU
    if (hipMemcpy(hostDataOut, deviceDataOut, (size/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Failed to get memory back\n");
    }

    // Print out data 
    //printData<<<blocks, threadsPerBlock>>>(size/2+1, deviceDataOut);
    hipDeviceSynchronize();

    // Copy to out
    for (int i = 0; i < size/2 + 1; i++) {
        out[i].r = hostDataOut[i].x;
        out[i].i = hostDataOut[i].y;
        //printf("Host: %f %f, Out: %f %f\n", hostDataOut[i].x, hostDataOut[i].y, out[i].r, out[i].i);
    }

    // Cleanup
    hipfftDestroy(plan);
    hipFree(deviceDataIn);
    hipFree(deviceDataOut);
    hipFree(deviceShortArray);

    return 0;
}
