#include "hip/hip_runtime.h"
#include <stdio.h>
#include <kiss_fftr.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__global__ void memAssign(int N, unsigned short* in, hipfftReal* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        out[index] = in[index];
        printf("CUDA: float: %f, short: %i\n", out[index], in[index]);
    }
}

__global__ void printData(int N, hipfftComplex* out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) 
        printf("CUDA: Output FFT %f %f\n", out[index].x, out[index].y);
}

void cudaTest() {
    printf("Inside Cuda file\n");
}

int cudaFFT(unsigned short* sample, int size, kiss_fft_cpx* out) {

    // Assign Variables
    hipfftHandle plan;
    hipfftReal* deviceDataIn;
    hipfftComplex* deviceDataOut;
    hipfftComplex* hostDataOut;
    unsigned short* deviceShortArray;

    // Malloc Variables 
    hipMalloc(&deviceDataIn, sizeof(hipfftReal) * size);
    hipMalloc(&deviceDataOut, sizeof(hipfftReal) * (size/2 + 1));
    hipMalloc(&deviceShortArray, sizeof(unsigned short) * size);
    hostDataOut = (hipfftComplex*)malloc(sizeof(hipfftComplex) * (size/2+1));
    if (hipGetLastError() != hipSuccess) {
        printf("Failed to allocate stuff on GPU\n");
        return 0;
    }

    // Copy memory over
     if (hipMemcpy(deviceShortArray, sample, size, hipMemcpyHostToDevice) != hipSuccess) {
         printf("Failed to copy shorts over\n");
         return 0;
     }

    // Run a Kernel to convert to the hipfftReal format
    int threadsPerBlock = 512;
    int blocks = (size + threadsPerBlock - 1)/threadsPerBlock;
    memAssign<<<blocks, threadsPerBlock>>>(size, deviceShortArray, deviceDataIn);
    hipDeviceSynchronize();

    // Now run the fft
    if (hipfftPlan1d(&plan, size, HIPFFT_R2C, 1) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - plan creation failed\n");
        return 0;
    }
    if (hipfftExecR2C(plan, deviceDataIn, deviceDataOut) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - execution of FFT failed\n");
        return 0;
    }
    if (hipDeviceSynchronize() != hipSuccess) {
        printf("Failed to sync\n");
        return 0;
    }
    // Get data back from GPU
    if (hipMemcpy(hostDataOut, deviceDataOut, size/2+1, hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Failed to get memory back\n");
    }

    // Print out data 
    printData<<<blocks, threadsPerBlock>>>(size/2+1, deviceDataOut);
    hipDeviceSynchronize();

    // Copy to out
    for (int i = 0; i < size/2 + 1; i++) {
        out[i].r = hostDataOut[i].x;
        out[i].i = hostDataOut[i].y;
    }

    // Cleanup
    hipfftDestroy(plan);
    hipFree(deviceDataIn);
    hipFree(deviceDataOut);
    hipFree(deviceShortArray);

    return 0;
}
