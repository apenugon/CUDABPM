#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "BeatCalculatorParallel.h"

#ifndef LIBINC
#define LIBINC
#include <mpg123.h>
#include <kiss_fftr.h>
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void differentiate_kernel(int size, unsigned short* array, hipfftReal* differentiated) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0 || index == size - 1) {
        differentiated[index] = array[index];
    }
    else if (index < size) {
        differentiated[index] = 44100 * (array[index+1]-array[index-1])/2;
    }
}

int BeatCalculatorParallel::cuda_detect_beat(char* s) {
    int max_freq = 4096;
    int sample_size = 2.2 * 2 * max_freq;
    int threadsPerBlock = 512;
    int blocks = (sample_size + threadsPerBlock - 1)/threadsPerBlock;

    // Load mp3
    unsigned short* sample = (unsigned short*)malloc(sizeof(unsigned short) * sample_size);
    readMP3(s, sample);

    // Step 2: Differentiate
    unsigned short* deviceSample;
    hipfftReal* deviceDifferentiatedSample;

    gpuErrchk( hipMalloc(&deviceSample, sizeof(unsigned short) * sample_size));
    gpuErrchk( hipMalloc(&deviceDifferentiatedSample, sizeof(hipfftReal) * sample_size));

    gpuErrchk( hipMemcpy(deviceSample, sample, sample_size * sizeof(unsigned short), hipMemcpyHostToDevice));

    differentiate_kernel<<<blocks, threadsPerBlock>>>(sample_size, deviceSample, deviceDifferentiatedSample);
    gpuErrchk( hipDeviceSynchronize());

    // Perform FFT
    hipfftHandle plan1D;
    hipfftComplex* deviceFFTOut;

    int out_size = sample_size/2 + 1;
    gpuErrchk( hipMalloc(&deviceFFTOut, sizeof(hipfftComplex) * out_size));
    
    if (hipfftPlan1d(&plan1D, sample_size, HIPFFT_R2C, 1) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - plan creation failed\n");
        return 0;
    }
    if (hipfftExecR2C(plan1D, deviceDifferentiatedSample, deviceFFTOut) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - execution of FFT failed\n");
        return 0;
    }

    gpuErrchk(hipFree(deviceSample));
    gpuErrchk(hipFree(deviceDifferentiatedSample));
    return 0;
}
