#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "BeatCalculatorParallel.h"

#ifndef LIBINC
#define LIBINC
#include <mpg123.h>
#include <kiss_fftr.h>
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void differentiate_kernel(int size, unsigned short* array, hipfftReal* differentiated) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0 || index == size - 1) {
        differentiated[index] = (hipfftReal)array[index];
    }
    else if (index < size) {
        differentiated[index] = 44100 * ((hipfftReal)array[index+1]-(hipfftReal)array[index-1])/2;
    }
}

//TODO: look up best way to reduce an array with CUDA
//      Currently use first thread in each block to reduce the array corresponding to that block, then return size N array
//      Once best found, make this function return single integer
<<<<<<< HEAD
__global__ void calculate_energy(hipfftComplex* sample, hipfftComplex* combs, int* tempEnergies, int * energies, int sample_size, int N) {
    int combIdx = blockIdx.x * sample_size;
||||||| merged common ancestors
__global__ void calculate_energy(hipfftComplex* sample, hipfftComplex* combs, int* tempEnergies, int * energies, int sample_size, int N) {
    int combIdx = blockIdx.x * blockDim.x;
=======
__global__ void calculate_energy(hipfftComplex* sample, hipfftComplex* combs, int* tempEnergies, int* energies, int sample_size, int N) {
    int combIdx = blockIdx.x * sample_size;
>>>>>>> some more changes
    int sampleIdx = threadIdx.x;

    if (sampleIdx < sample_size) {
      int a = sample[sampleIdx].x * combs[combIdx + sampleIdx].x - sample[sampleIdx].y * combs[combIdx + sampleIdx].y;
      int b = sample[sampleIdx].x * combs[combIdx + sampleIdx].y + sample[sampleIdx].y * combs[combIdx + sampleIdx].x;
      tempEnergies[combIdx + sampleIdx] = a * a + b * b;
    }

    __syncthreads();

    if (sampleIdx == 0) {
      int energy = 0;
      for (int i=0; i < sample_size; i++) {
        energy += tempEnergies[combIdx+i];
      }
      energies[blockIdx.x] = energy;
    }
}

//TODO: write kernel function to do this
void generateCombs(int BPM_init, int N, int size, int AmpMax, hipfftReal* hostDataIn) {
    for(int i = 0; i < N; i++) {
      int BPM = BPM_init + i*5;
      int Ti = 60 * 44100/BPM;
      int start = size * i; //compute offset for this comfilter

      for(int k = 0; k < size; k+=2) {
        if (k % Ti == 0) {
          hostDataIn[start+k] = AmpMax;
          hostDataIn[start+k+1] = AmpMax;
        }
        else {
          hostDataIn[k] = 0;
          hostDataIn[k+1] = 0;
        }
      }
    }
}

void combFilterFFT(int BPM_init, int BPM_final, int N, int fft_input_size, hipfftComplex* deviceDataOut) {

    // Assign Variables
    hipfftHandle plan;
    hipfftReal* deviceDataIn, *hostDataIn;


    int AmpMax = 65535;
     
    // Malloc Variables 
    gpuErrchk( hipMalloc(&deviceDataIn, sizeof(hipfftReal) * fft_input_size * N) );

    hostDataIn = (hipfftReal*)malloc(sizeof(hipfftReal) * fft_input_size * N);
    
    //Generate all Combs
    generateCombs(BPM_init, N, fft_input_size, AmpMax, hostDataIn);

    int n[1] = {fft_input_size};

    gpuErrchk( hipMemcpy(deviceDataIn, hostDataIn, fft_input_size * N * sizeof(hipfftReal), hipMemcpyHostToDevice) );

    // Now run the fft
    if (hipfftPlanMany(&plan, 1, n, NULL, 1, fft_input_size, NULL, 1, fft_input_size, HIPFFT_R2C, N) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - plan creation failed\n");
        exit(-1);
    }
    if (hipfftExecR2C(plan, deviceDataIn, deviceDataOut) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - execution of FFT failed\n");
        exit(-1);
    }

    gpuErrchk( hipDeviceSynchronize() );

    // Cleanup
    if (hipfftDestroy(plan) != HIPFFT_SUCCESS) {
      printf("CUFFT Error - plan destruction failed\n");
      exit(-1);
    }

    gpuErrchk( hipFree(deviceDataIn) );
    free(hostDataIn);

    return;
}

int combFilterAnalysis(hipfftComplex* sample, hipfftComplex* combs, int out_size, int N) {
    //Launch a kernel to calculate the instant energy at position $k$ in the filtered sample, for all k, for all N filters

    //Run Kernel to determine energies
    int *tempEnergies, *deviceEnergies, *hostEnergies;
    gpuErrchk( hipMalloc(&tempEnergies, sizeof(int) * out_size * N) );
    gpuErrchk( hipMalloc(&deviceEnergies, sizeof(int) * N) );

    hostEnergies = (int*)malloc(N * sizeof(int));

    const int blocks = N; //want a block for each comb

    const int tpb = 512;

    calculate_energy<<<blocks, tpb>>>(sample, combs, tempEnergies, deviceEnergies, out_size, N);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //free temp array
    gpuErrchk( hipFree(tempEnergies) );
    
    //Loop through final array to find the best one
    gpuErrchk( hipMemcpy(hostEnergies, deviceEnergies, sizeof(int) * N, hipMemcpyDeviceToHost) );

    //Calculate max of 
    int max = -1;
    int index = -1;
    for (int i = 0; i < 30; i++) {
        if (hostEnergies[i] > max) {
            max = hostEnergies[i];
            index = i;
        }
    }
    
    gpuErrchk( hipFree(deviceEnergies) );

    return 60 + index * 5;
}

int BeatCalculatorParallel::cuda_detect_beat(char* s) {
    int max_freq = 4096;
    int sample_size = 2.2 * 2 * max_freq;
    int threadsPerBlock = 512;
    int blocks = (sample_size + threadsPerBlock - 1)/threadsPerBlock;

    // Load mp3
    unsigned short* sample = (unsigned short*)malloc(sizeof(unsigned short) * sample_size);
    readMP3(s, sample);

    // Step 2: Differentiate
    unsigned short* deviceSample;
    hipfftReal* deviceDifferentiatedSample;

    gpuErrchk( hipMalloc(&deviceSample, sizeof(unsigned short) * sample_size));
    gpuErrchk( hipMalloc(&deviceDifferentiatedSample, sizeof(hipfftReal) * sample_size));

    gpuErrchk( hipMemcpy(deviceSample, sample, sample_size * sizeof(unsigned short), hipMemcpyHostToDevice));

    //free sample array on host
    free(sample);

    //differentiate sample on device
    differentiate_kernel<<<blocks, threadsPerBlock>>>(sample_size, deviceSample, deviceDifferentiatedSample);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Perform FFT
    hipfftHandle plan1D;
    hipfftComplex* deviceFFTOut;

    int out_size = sample_size/2 + 1;
    gpuErrchk( hipMalloc(&deviceFFTOut, sizeof(hipfftComplex) * out_size));
    
    if (hipfftPlan1d(&plan1D, sample_size, HIPFFT_R2C, 1) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - plan creation failed\n");
        return 0;
    }
    if (hipfftExecR2C(plan1D, deviceDifferentiatedSample, deviceFFTOut) != HIPFFT_SUCCESS) {
        printf("CUFFT Error - execution of FFT failed\n");
        return 0;
    }
    
    //free diff'd sample (we don't need it anymore)
    gpuErrchk( hipFree(deviceDifferentiatedSample) );

    //Create Combs + FFT them
    hipfftComplex* combFFTOut;
    int BPM_init = 60;
    int BPM_final = 210;
    int N = (BPM_final - BPM_init)/5;
    gpuErrchk( hipMalloc(&combFFTOut, sizeof(hipfftComplex) * out_size * N) );
    
    combFilterFFT(BPM_init, BPM_final, N, sample_size, combFFTOut);

    //perform analysis
    int BPM = combFilterAnalysis(deviceFFTOut, combFFTOut, out_size, N);

    gpuErrchk(hipFree(combFFTOut));
    gpuErrchk(hipFree(deviceFFTOut));
    
    return BPM;
}
